#include "hip/hip_runtime.h"
/**
 * Copyright 2015 Erik Zenker, Carlchristian Eckert, Marius Melzer
 *
 * This file is part of HASEonGPU
 *
 * HASEonGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * HASEonGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with HASEonGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */


/**
 * @mainpage HASEonGPU - High performance Amplified Spontaneous EmissioN on GPU
 *
 * Project with HZDR for porting their ASE-code to a GPU cluster.
 *
 * @author Erik Zenker, Carlchristian Eckert, Marius Melzer
 */

// Libraries
#include <assert.h> /* assert */
#include <string> /* string */
#include <vector> /* vector */
#include <stdlib.h> /* atoi */
#include <pthread.h> /* pthread_t, pthread_join */
#include <algorithm> /* std::max */
#include <numeric> /* accumulate*/
#include <stdexcept>

// Boost stuff
#include <boost/filesystem.hpp> /* fs::path */
namespace fs = boost::filesystem;

// User header files
#include <calc_phi_ase.hpp>
#include <calc_phi_ase_threaded.hpp>
#include <calc_phi_ase_mpi.hpp>
#include <calc_phi_ase_graybat.hpp>
#include <parser.hpp> /* DeviceMode, ParallelMode */
#include <write_to_vtk.hpp>
#include <write_matlab_output.hpp>
#include <for_loops_clad.hpp>
#include <cudachecks.hpp>
#include <mesh.hpp>
#include <cuda_utils.hpp> /* getFreeDevices */
#include <logging.hpp>
#include <ray_histogram.hpp>
#include <types.hpp>

// default without V_DEBUG
unsigned verbosity = V_ERROR | V_INFO | V_WARNING | V_PROGRESS | V_STAT; // extern through logging.hpp

/** 
 * @brief Calculates dndt ASE from phi ASE values
 * 
 * @param mesh needed for some constants
 * @param sigmaA absorption
 * @param sigmaE emission
 * @param phiAse results from calcPhiAse
 * @param sample_i index of sample point
 * @return dndtAse
 *
 */
double calcDndtAse(const Mesh& mesh, const double sigmaA, const double sigmaE, const float phiAse, const unsigned sample_i){
    double gain_local = mesh.nTot * mesh.betaCells[sample_i] * (sigmaE + sigmaA) - double(mesh.nTot * sigmaA);
    return gain_local * phiAse / mesh.crystalTFluo;
}

int main(int argc, char **argv){


    ExperimentParameters experiment;
    ComputeParameters    compute;
    Result               result;
    Mesh                 mesh;
    
    parse(argc, argv, experiment, compute, mesh, result);
  
    // Run Experiment
    // std::vector<pthread_t> threadIds(maxGpus, 0);
    // std::vector<float> runtimes(maxGpus, 0);

    switch(compute.deviceMode){
    case NO_DEVICE_MODE:
	dout(V_ERROR) << "No valid device-mode!" << std::endl;
	exit(1);

	// case CPU_DEVICE_MODE: //Possibly deprecated!
	//   // TODO: make available for MPI?
	//   runtime = forLoopsClad( &dndtAse,
	//       minRaysPerSample,
	//       &meshs[0],
	//       meshs[0].betaCells,
	//       meshs[0].nTot,
	//       sigmaA.at(0),
	//       sigmaE.at(0),
	//       meshs[0].numberOfPoints,
	//       meshs[0].numberOfTriangles,
	//       meshs[0].numberOfLevels,
	//       meshs[0].thickness,
	//       meshs[0].crystalTFluo);
	//   runmode = "CPU Mode single threaded";
	//   break;

    case GPU_DEVICE_MODE:
	switch(compute.parallelMode){

	    // // TODO: Replace completly by MPI
	    // case THREADED_PARALLEL_MODE:
	    //   for(unsigned gpu_i = 0; gpu_i < maxGpus; ++gpu_i){
	    //     const unsigned samplesPerNode = maxSampleRange-minSampleRange+1;
	    //     const float samplePerGpu = samplesPerNode / (float) maxGpus;
	    //     unsigned minSample_i = gpu_i * samplePerGpu;
	    //     unsigned maxSample_i = min((float)samplesPerNode, (gpu_i + 1) * samplePerGpu);

	    //     minSample_i += minSampleRange;
	    //     maxSample_i += minSampleRange; 

	    //     threadIds[gpu_i] = calcPhiAseThreaded( minRaysPerSample,
	    //         maxRaysPerSample,
	    //         maxRepetitions,
	    //         meshs[gpu_i],
	    //         sigmaAInterpolated,
	    //         sigmaEInterpolated,
	    //         mseThreshold,
	    //         useReflections,
	    //         phiAse, 
	    //         mse, 
	    //         totalRays,
	    //         devices.at(gpu_i),
	    //         minSample_i,
	    //         maxSample_i,
	    //         runtimes.at(gpu_i)
	    //         );
	    //   }
	    //   joinAll(threadIds);
	    //   usedGpus = maxGpus;
	    //   for(std::vector<float>::iterator it = runtimes.begin(); it != runtimes.end(); ++it){
	    //     runtime = max(*it, runtime);
	    //   }
	    //   hipDeviceReset();      
	    //   runmode="GPU mode Threaded";
	    //   break;

	    // case MPI_PARALLEL_MODE:
	    //   usedGpus = calcPhiAseMPI( minRaysPerSample,
	    //       maxRaysPerSample,
	    //       maxRepetitions,
	    //       meshs[0],
	    //       sigmaAInterpolated,
	    //       sigmaEInterpolated,
	    //       mseThreshold,
	    //       useReflections,
	    //       phiAse,
	    //       mse,
	    //       totalRays,
	    //       devices.at(0)
	    //       );
	    //   runmode = "GPU mode MPI";
	    //   break;
	  
	case GRAYBAT_PARALLEL_MODE:
	    //usedGpus =
	    calcPhiAseGrayBat( experiment,
			       compute,
			       mesh,
			       result );
	    //runmode = "GPU mode GrayBat";
	    break;

        default:
	    dout(V_ERROR) << "No valid parallel-mode for GPU!" << std::endl;
	    exit(1);
	}

    }


    // // Print Solution
    // if(verbosity & V_DEBUG){
    //   for(unsigned sample_i = 0; sample_i < meshs[0].numberOfSamples; ++sample_i){
    //     dndtAse.at(sample_i) = calcDndtAse(meshs[0], maxSigmaA, maxSigmaE, phiAse.at(sample_i), sample_i);
    //     if(sample_i <=10)
    // 	dout(V_DEBUG) << "Dndt ASE[" << sample_i << "]: " << dndtAse.at(sample_i) << " " << mse.at(sample_i) << std::endl;
    //   }
    //   for(unsigned sample_i = 0; sample_i < meshs[0].numberOfSamples; ++sample_i){
    //     dout(V_DEBUG) << "PHI ASE[" << sample_i << "]: " << phiAse.at(sample_i) << " " << mse.at(sample_i) <<std::endl;
    //     if(sample_i >= 10) break;
    //   }
    // }

    // // Write experiment data
    // // output folder has to be the same as TMP_FOLDER in the calling MatLab script
    // writeMatlabOutput(outputPath,
    // 		    phiAse,
    // 		    totalRays,
    // 		    mse,
    // 		    meshs[0].numberOfSamples,
    // 		    meshs[0].numberOfLevels);

    // // Write solution to vtk files
    // if(writeVtk){
    //   std::vector<double> tmpPhiAse(phiAse.begin(), phiAse.end());
    //   std::vector<double> tmpTotalRays(totalRays.begin(), totalRays.end());

    //   writePointsToVtk(meshs[0], dndtAse, outputPath /= "vtk/dndt", minRaysPerSample, maxRaysPerSample, mseThreshold, useReflections, runtime);
    //   writePointsToVtk(meshs[0], tmpPhiAse, outputPath /= "vtk/phiase", minRaysPerSample, maxRaysPerSample, mseThreshold, useReflections, runtime);
    //   writePointsToVtk(meshs[0], mse, outputPath /= "vtk/mse", minRaysPerSample, maxRaysPerSample, mseThreshold, useReflections, runtime);
    //   writePointsToVtk(meshs[0], tmpTotalRays, outputPath /= "vtk/total_rays", minRaysPerSample, maxRaysPerSample, mseThreshold, useReflections, runtime);
    // }

    // // Print statistics
    // if(verbosity & V_STAT){
    //   for(std::vector<double>::iterator it = mse.begin(); it != mse.end(); ++it){
    //     maxMSE = max(maxMSE, *it);
    //     avgMSE += *it;
    //     if(*it >= mseThreshold)
    //       highMSE++;
    //   }
    //   avgMSE /= mse.size();

    //   try{ std::cout.imbue(std::locale("")); }
    //   catch(std::runtime_error e){}

    //   dout(V_STAT | V_NOLABEL) << std::endl;
    //   dout(V_STAT) << "=== Statistics ===" << std::endl;
    //   dout(V_STAT) << "Runmode           : " << runmode << std::endl;
    //   dout(V_STAT) << "Prisms            : " << (int) meshs[0].numberOfPrisms << std::endl;
    //   dout(V_STAT) << "Samples           : " << (int) dndtAse.size() << std::endl;
    //   dout(V_STAT) << "RaysPerSample     : " << minRaysPerSample;
    //   if(maxRaysPerSample > minRaysPerSample) { dout(V_STAT | V_NOLABEL) << " - " << maxRaysPerSample << " (adaptive)"; }
    //   dout(V_STAT | V_NOLABEL) << std::endl;
    //   dout(V_STAT) << "sum(totalRays)    : " << std::accumulate(totalRays.begin(), totalRays.end(), 0.) << std::endl;
    //   dout(V_STAT) << "MSE threshold     : " << mseThreshold << std::endl;
    //   dout(V_STAT) << "Wavelength        : " << sigmaA.size() << std::endl;
    //   dout(V_STAT) << "int. Wavelength   : " << sigmaAInterpolated.size() << std::endl;
    //   dout(V_STAT) << "max. MSE          : " << maxMSE << std::endl;
    //   dout(V_STAT) << "avg. MSE          : " << avgMSE << std::endl;
    //   dout(V_STAT) << "too high MSE      : " << highMSE << std::endl;
    //   dout(V_STAT) << "Nr of GPUs        : " << usedGpus << std::endl;
    //   dout(V_STAT) << "Runtime           : " << difftime(time(0),starttime) << "s" << std::endl;
    //   dout(V_STAT) << std::endl;
    //   if(maxRaysPerSample > minRaysPerSample){
    //     dout(V_STAT) << "=== Sampling resolution as Histogram ===" << std::endl;
    //     ray_histogram(totalRays,maxRaysPerSample,mseThreshold,mse);
    //   }
    //   dout(V_STAT) << std::endl;

    // }
  
    return 0;

}
