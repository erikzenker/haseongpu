#include "hip/hip_runtime.h"
/**
 * Copyright 2013 Erik Zenker, Carlchristian Eckert, Marius Melzer
 *
 * This file is part of HASEonGPU
 *
 * HASEonGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * HASEonGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with HASEonGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */


#include <cassert> /* assert */

#include <mesh.hpp>
#include <geometry.hpp> /* generateRay */
#include <propagate_ray.hpp> /* propagateRay */
#include <reflection.hpp> /* ReflectionPlane */

#include <hiprand/hiprand_kernel.h> /*hiprand_uniform*/

/**
 * @brief get the offset for accessing indicesOfPrisms and numberOfReflectionSlices (slow!).
 *
 * @param blockOffset shared memory location that holds the offset for the whole block (4 warps)
 * @param raysPerSample number of raysPerSample (can be any number higher than raysPerSample/warpsize)
 * @param globalOffsetMultiplicator is incremented by 1 each time a warp asks for a new workload
 * @return an unused offset in the global arrays indicesOfPrisms/numberOfReflectionSlices
 *
 */
__device__ unsigned getRayNumberWarpbased(unsigned* blockOffset,unsigned raysPerSample, unsigned *globalOffsetMultiplicator){
	// if this is warpID 0
	if((threadIdx.x &31) == 0){
		//get a new offset for the warp (threadId % 32)
		blockOffset[(threadIdx.x>>5)] = atomicInc(globalOffsetMultiplicator,raysPerSample);
	}
	__syncthreads();

	// multiply blockoffset by 32 (size of warp)
	return (threadIdx.x &31) + (blockOffset[(threadIdx.x>>5)] <<5) ;

}

/**
 * @brief get the offset for accessing indicesOfPrisms and numberOfReflectionSlices.
 *        Warning: works only for a blocksize of 128 threads!
 *
 * @param blockOffset shared memory location that holds the offset for the whole block
 * @param raysPerSample number of raysPerSample (can be any number higher than raysPerSample/blocksize)
 * @param globalOffsetMultiplicator is incremented by 1 each time a block asks for a new workload
 * @return an unused offset in the global arrays indicesOfPrisms/numberOfReflectionSlices
 *
 */
__device__ unsigned getRayNumberBlockbased(unsigned* blockOffset,unsigned raysPerSample,unsigned *globalOffsetMultiplicator){
	// The first thread in the threadblock increases the globalOffsetMultiplicator (without real limit)

    	__syncthreads();
    
	if(threadIdx.x == 0){
		//blockOffset is the new value of the globalOffsetMultiplicator
		blockOffset[0] = atomicAdd(globalOffsetMultiplicator,1);
	}
	__syncthreads();

	//multiply blockOffset by 128 (size of the threadblock) 
	return threadIdx.x + (blockOffset[0] * 128) ;
}

/**
 * @brief get a random number from [0..length)
 *
 * @param length the maximum number to return (exclusive)
 * @param globalState State for random number generation (mersenne twister).
 *                    The state need to be initialized before. See
 *                    http://www.math.sci.hiroshima-u.ac.jp/~m-mat/MT/MTGP/
 *                    for more information.
 *
 * @return a random number
 *
 */
__device__ __inline__ unsigned genRndSigmas(unsigned length, hiprandStateXORWOW_t &rndState) {
  return unsigned(hiprand_uniform(&rndState)*(length-1));
}

__global__ void calcSampleGainSumWithReflection(
						const Mesh mesh, 
						const unsigned* indicesOfPrisms, 
						const unsigned* numberOfReflectionSlices,
						const double* importance,
						const unsigned raysPerSample,
						float *gainSum, 
						float *gainSumSquare,
						const unsigned sample_i,
						const double *sigmaA, 
						const double *sigmaE,
						const unsigned maxInterpolation,
						unsigned *globalOffsetMultiplicator
						) {

    unsigned const subsequence = 0;
    unsigned const offset = 0;
    unsigned const seed = 4321;
  
    hiprandStateXORWOW_t rndState;
  
    hiprand_init(seed,
		subsequence,
		offset,
		&rndState);

  int rayNumber = 0;
  double gainSumTemp = 0;
  double gainSumSquareTemp = 0;
  Point samplePoint = mesh.getSamplePoint(sample_i);
  __shared__ unsigned blockOffset[4]; // 4 in case of warp-based raynumber

  if(threadIdx.x == 0){
      blockOffset[0] = 0;
  }
  
  __syncthreads();
  
  // One thread can compute multiple rays

  while(blockOffset[0] * 128 < raysPerSample){

	// the whole block gets a new offset (==workload)
	rayNumber = getRayNumberBlockbased(blockOffset,raysPerSample,globalOffsetMultiplicator);
	if(rayNumber < raysPerSample) {

	    // Get triangle/prism to start ray from
	    unsigned startPrism             = indicesOfPrisms[rayNumber];
	    unsigned reflection_i           = numberOfReflectionSlices[rayNumber];
	    unsigned reflections            = (reflection_i + 1) / 2;
	    ReflectionPlane reflectionPlane = (reflection_i % 2 == 0) ? BOTTOM_REFLECTION : TOP_REFLECTION;
	    unsigned startLevel             = startPrism / mesh.numberOfTriangles;
	    unsigned startTriangle          = startPrism - (mesh.numberOfTriangles * startLevel);
	    unsigned reflectionOffset       = reflection_i * mesh.numberOfPrisms;
	    Point startPoint                = mesh.genRndPoint(startTriangle, startLevel, rndState);
	
	    //get a random index in the wavelength array
	    unsigned sigma_i                = 0;//genRndSigmas(maxInterpolation, rndState);

	    // Calculate reflections as different ray propagations
	    double gain    = propagateRayWithReflection(startPoint, samplePoint, reflections, reflectionPlane, startLevel, startTriangle, mesh, sigmaA[sigma_i], sigmaE[sigma_i]);

	    // include the stimulus from the starting prism and the importance of that ray
	    gain          *= mesh.getBetaVolume(startPrism) * importance[startPrism + reflectionOffset];
    
	    assert(!isnan(mesh.getBetaVolume(startPrism)));
	    assert(!isnan(importance[startPrism + reflectionOffset]));
	    assert(!isnan(gain));

	    gainSumTemp       += gain;
	    gainSumSquareTemp += gain * gain;
	}


  }
  atomicAdd(&(gainSum[0]), float(gainSumTemp));
  atomicAdd(&(gainSumSquare[0]), float(gainSumSquareTemp));

}

__global__ void calcSampleGainSum(
				  const Mesh mesh, 
				  const unsigned* indicesOfPrisms, 
				  const double* importance,
				  const unsigned raysPerSample,
				  float *gainSum, 
				  float *gainSumSquare,
				  const unsigned sample_i,
				  const double* sigmaA, 
				  const double* sigmaE,
				  const unsigned lambdaResolution,
				  unsigned *globalOffsetMultiplicator
				  ) {
    
    int rayNumber = 0; 
    double gainSumTemp = 0;
    double gainSumSquareTemp = 0;
    Point samplePoint = mesh.getSamplePoint(sample_i);
  
    __shared__ unsigned blockOffset[4]; // 4 in case of warp-based raynumber

    if(threadIdx.x == 0){
	blockOffset[0] = 0;
    }
  
    __syncthreads();
  
    // One thread can compute multiple rays
    while(blockOffset[0] * 128 < raysPerSample){
	// the whole block gets a new offset (==workload)
	rayNumber = getRayNumberBlockbased(blockOffset,raysPerSample,globalOffsetMultiplicator);
	if(rayNumber < raysPerSample) {

	    // Get triangle/prism to start ray from
	    unsigned startPrism             = indicesOfPrisms[rayNumber];
	    unsigned startLevel             = startPrism/mesh.numberOfTriangles;
	    unsigned startTriangle          = startPrism - (mesh.numberOfTriangles * startLevel);
	    Point startPoint                = mesh.getCenterPoint(startTriangle, startLevel);//mesh.genRndPoint(startTriangle, startLevel, rndState);
	    Ray ray                         = generateRay(startPoint, samplePoint);

	    // get a random index in the wavelength array
	    unsigned sigma_i                = 0;//genRndSigmas(lambdaResolution, rndState);
	    assert(sigma_i < lambdaResolution);

	    // calculate the gain for the whole ray at once
	    double gain    = propagateRay(ray, &startLevel, &startTriangle, mesh, sigmaA[sigma_i], sigmaE[sigma_i]);
	    gain          /= ray.length * ray.length; // important, since usually done in the reflection device function

	    // include the stimulus from the starting prism and the importance of that ray
	    gain          *= mesh.getBetaVolume(startPrism) * importance[startPrism];

	    gainSumTemp       += gain;
	    gainSumSquareTemp += gain * gain;

	}

    }
    atomicAdd(&(gainSum[0]), float(gainSumTemp));
    atomicAdd(&(gainSumSquare[0]), float(gainSumSquareTemp));

}
